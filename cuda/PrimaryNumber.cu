
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <memory.h>
#include <stdlib.h>

// these numbers can not be larger, it is not because of GPU global memory size limit(for example:flag size),
// maybe it is limits by other level memory size.
const unsigned int blockNum = 1024;
const unsigned int threadNum = 1024;

__global__ void isDivisible(unsigned long value, unsigned long start, unsigned char *flag)
{
	//int threadid = threadIdx.x + blockNum * blockIdx.x;
	int threadid = blockDim.x * blockIdx.x + threadIdx.x;
	if ((value % (start + threadid)) == 0)
	{
		flag[threadid] = 1;
	}
}

int checkPrimary(unsigned long value)
{
	hipError_t cudaStatus;
	unsigned char * dev_flag;
	unsigned char * host_flag;
	
	const  int size =  blockNum * threadNum;

	host_flag = (unsigned char*)malloc(size * sizeof(unsigned char));
	if (host_flag == NULL)
	{
		fprintf(stderr, "malloc failed!");
		return -1;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_flag, size * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	for (;; value++)
	{
		hipMemset(dev_flag, 0, size * sizeof(unsigned char));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemset failed!");
			goto Error;
		}
		bool isPrimary = true;
		for (unsigned long i = 2; i <= (value / 2); i += size)
		{
			isDivisible <<<blockNum, threadNum >>> (value, i, dev_flag);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "isDivisible launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}
			// cudaDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching isDivisible:%s!\n", cudaStatus, hipGetErrorString(cudaStatus));
				goto Error;
			}

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(host_flag, dev_flag, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				goto Error;
			}
			
			for (int j = 0; j < size; ++j)
			{
				if (host_flag[j])
				{
					isPrimary = false;
					//printf("[%llu] [%llu]\n",value, i + j);
					break;
				}
			}
			if (isPrimary == false)
			{
				break;
			}

		}
		if (isPrimary)
		{
			printf("%llu is a primary number\n", value);
		}

	}
	


Error:

	hipFree(dev_flag);
	free(host_flag);
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}
	return 0;

}
